#include "hip/hip_runtime.h"
#include "spatial.hpp"
#include "boundary.hpp"
#include "integrate.hpp"
#include <cstdlib>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>

#define _errorcheck
#define substep 1
#define tol 1e-4
// #define _debug
// #define _verbose

// __global__ static void printPositions(sphParticle *d_particles_, uint32_t n_particles) {  
//  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
//  if (idx >= n_particles) return;

#ifdef _verbose
    printf("idx: %u : <%f,%f,%f>\n",
      idx, 
      d_particles_[idx].position[0],
      d_particles_[idx].position[1],
      d_particles_[idx].position[2]
    );
#endif
// }

__global__ static void setAccumulators(particleContainer *d_particleContainer_, uint32_t n_particles) {
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n_particles) return;

  // Quickly reset all acculated values from previous iteration
  d_particleContainer_->densities[idx]  = tol;
  d_particleContainer_->pressures[idx]  = 0.0;
  for (int i = 0; i < 3; ++i) {

    uint32_t co = idx + i * n_particles;

    d_particleContainer_->pressure_forces[co]  = 0.0;
    d_particleContainer_->viscosity_forces[co] = 0.0;
  }
}

/* Copies positions into contiguous device buffer */ 
__global__ static void updateHostBuffer(
  particleContainer *d_particleContainer_,
  float *u_positions,
  float *u_densities,
  uint32_t n_particles
)
{
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n_particles) return;

  u_densities[idx] = d_particleContainer_->densities[idx];
  for (int i = 0; i < 3; ++i) {
    u_positions[idx * 3 + i] = d_particleContainer_->positions[idx + i * n_particles];
#ifdef _debug  
    printf("Position %d for idx: %u : %f\n",
      i, idx, d_positions[idx * 3 + i]
    );
#endif
  }
}

/* Generates positions from particle array */
__host__ void particleIterator(
  spatialLookupTable *d_lookup_,
  particleContainer *d_particleContainer_, 
  float **u_positions,
  float **u_densities,
  std::vector<float> container,
  uint32_t n_particles,
  uint32_t n_partitions,
  const float h
)
{
  uint32_t threadsPerBlock = 256;
  uint32_t gridSize = (n_particles + threadsPerBlock - 1) / threadsPerBlock;
  uint32_t verletGrid = (gridSize < 30) ? 30 : gridSize;
  // threadsPerBlock ~= (n_particles - 1) / (verletGrid - 1) -> findSquare()
  uint32_t verletTPB = findSquare((n_particles - 1) / (verletGrid - 1));

#ifdef _verbose  
  printPositions<<<gridSize, threadsPerBlock>>>(d_particles_, n_particles);
#endif
  // Update bounds if container or particle count have changed
  updateBounds(d_lookup_, d_particleContainer_, container, n_particles, h);

  struct Container boundary = {
    .lower = {0.0, 0.0, 0.0},
    .upper = {container[0], container[1], container[2]}
  };

  setAccumulators<<<gridSize, threadsPerBlock>>>(d_particleContainer_, n_particles);

  // Set lookup spatialLookupTable for new positions 
  uint32_t padded_size = findSquare(n_particles);
  hostFillTable(d_lookup_, d_particleContainer_, n_partitions, n_particles, padded_size, h);

  // Checking boundary conditions is breaking the particles positions...
  
  // Only one static boundary for now (the container itself)
  for (int i = 0; i < substep; ++i) { 
    callToBoundaryConditions(boundary, d_particleContainer_, n_particles, n_partitions, h);
  }
#ifdef _verbose
  printPositions<<<gridSize, threadsPerBlock>>>(d_particles_, n_particles);
#endif
  firstVerletKernel<<<verletGrid, verletTPB>>>(d_particleContainer_, n_particles);
  hipDeviceSynchronize();
#ifdef _errorcheck
  hipError_t verletErr = hipGetLastError();
  if (verletErr != hipSuccess) {
    std::cerr << "Verlet 1 Error: " << hipGetErrorString(verletErr) << '\n';
    exit(EXIT_FAILURE);
  }
#endif

  // Convert upper bounds to the max id for each direction
  static uint32_t *containerCount;
  if (containerCount == NULL/* || anyChange == true*/) {
    hipMallocManaged(&containerCount, 3 * sizeof(uint32_t));
    for (int i = 0; i < 3; ++i) {
      containerCount[i] = static_cast<uint32_t>(floor(container[i] / h));
    }
  };

#ifdef _debug
  for (int i = 0; i < 3; ++i) {
    std::cout << i << ": " << containerCount[i] << '\n';
  }
#endif
  // Calls the neighbor search
  callToNeighborSearch(
    d_lookup_,
    d_particleContainer_,
    n_partitions,
    n_particles,
    containerCount,
    h
  );

  // Second verlet pass with new force values
  secondVerletKernel<<<verletGrid, verletTPB>>>(d_particleContainer_, n_particles);
#ifdef _errorcheck
  verletErr = hipGetLastError();
  if (verletErr != hipSuccess) {
    std::cerr << "Verlet 2 Launch Error: " << hipGetErrorString(verletErr) << '\n';
    exit(EXIT_FAILURE);
  }
#endif

  hipDeviceSynchronize();
#ifdef _errorcheck
  verletErr = hipGetLastError();
  if (verletErr != hipSuccess) {
    std::cerr << "Verlet 2 Sync Error: " << hipGetErrorString(verletErr) << '\n';
    exit(EXIT_FAILURE);
  }
#endif
#ifdef _debug
  std::cout << "Completed Position Set\n";
#endif
  // If first iteration create managed malloc call
  if ((*u_positions) == NULL) {
    hipMallocManaged(u_positions, n_particles * sizeof(float) * 3);
  }

  if ((*u_densities) == NULL) {
    hipMallocManaged(u_densities, n_particles * sizeof(float));
  }

  // Creates single contiguous bufr of floats (for positions)
  updateHostBuffer<<<gridSize, threadsPerBlock>>>(d_particleContainer_, (*u_positions), (*u_densities), n_particles);

#ifdef _errorcheck 
  hipError_t launchErr = hipGetLastError();
  if (launchErr != hipSuccess) {
    std::cerr << "Launch Error: " << hipGetErrorString(launchErr) << '\n';
    exit(EXIT_FAILURE);
  }
#endif

  hipDeviceSynchronize();
#ifdef _errorcheck
  hipError_t syncErr = hipGetLastError();
  if (syncErr != hipSuccess) {
    std::cerr << "Sync Error: " << hipGetErrorString(syncErr) << '\n';
    exit(EXIT_FAILURE);
  }
#endif

#ifdef _debug
  std::cout << "Position Buffer created\n";
#endif

#ifdef _debug
  // fill 2d vector of positions
  for (uint32_t i = 0; i < n_particles; ++i) {
    for (int j = 0; j < 3; ++j) {
      host_positions[i][j] = host_vec[i * 3 + j];
      std::cout << "Position[i][j]: " << host_positions[i][j] << '\n';
    }
  }
#endif
}
