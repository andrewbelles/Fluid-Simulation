#include "hip/hip_runtime.h"
#include "iterate.hpp"
#include "integrate.hpp"

#define tol 1e-4

__global__ static void setAccumulators(particleContainer *d_objs_, uint32_t n_particles) {
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n_particles) return;

  // Quickly reset all acculated values from previous iteration
  d_objs_->densities[idx]  = tol;
  d_objs_->pressures[idx]  = 0.0;
  for (int i = 0; i < 3; ++i) {

    uint32_t co = idx + i * n_particles;

    d_objs_->pressure_forces[co]  = 0.0;
    d_objs_->viscosity_forces[co] = 0.0;
  }
}

/* Copies positions into contiguous device buffer */ 
__global__ static void updateHostBuffer(
  particleContainer *d_objs_,
  float *u_positions,
  float *u_densities,
  uint32_t n_particles
)
{
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n_particles) return;

  u_densities[idx] = d_objs_->densities[idx];
  for (int i = 0; i < 3; ++i) {
    u_positions[idx * 3 + i] = d_objs_->positions[idx + i * n_particles];
  }
}

/* Generates positions from particle array */

__host__ void particleIterator(
  neighborList *list,
  particleContainer *d_objs_,
  uint32_t *list_size,
  float **u_positions,
  float **u_densities,
  Lookup *d_lookup_,
  std::vector<float> container,
  uint32_t n_particles,
  uint32_t n_partitions, 
  float h
) {
  static uint32_t blocks = 0, threads = 0;
  hipError_t err;
  setGridSize(&blocks, &threads, n_particles);
#ifdef __debug
  std::cout << "Grid Set\n";
#endif

  // Update bounds if container or particle count have changed
  updateBounds(d_lookup_, d_objs_, container, n_particles, h);
#ifdef __debug
  std::cout << "Updated Bounds\n";  
#endif
  struct Container boundary = {
    .lower = {0.0, 0.0, 0.0},
    .upper = {container[0], container[1], container[2]}
  };

  setAccumulators<<<blocks, threads>>>(d_objs_, n_particles);
#ifdef __debug
  std::cout << "Set Accumulators\n";
#endif
  // Set lookup Lookup for new positions 
  uint32_t padded_size = findSquare(n_particles);
  hostFillTable(d_lookup_, d_objs_, n_partitions, n_particles, padded_size, h);
#ifdef __debug
  std::cout << "Filled Table\n";
#endif
  // Checking boundary conditions is breaking the particles positions...
  
  // Only one static boundary for now (the container itself)
  callToBoundaryConditions(boundary, d_objs_, n_particles, n_partitions, h);
#ifdef __debug 
  std::cout << "Enforced Boundary\n";
#endif
  // Launch the first half of verlet integration that doesn't require the next step of forces
  firstVerletKernel<<<blocks, threads>>>(d_objs_, n_particles);
  hipDeviceSynchronize();
#ifdef __debug
  std::cout << "First Verlet Pass\n";
#endif
  // Convert upper bounds to the max id for each direction (static container)
  static uint32_t *containerCount;
  if (containerCount == NULL) {
    hipMallocManaged(&containerCount, 3 * sizeof(uint32_t));
    for (int i = 0; i < 3; ++i) {
      containerCount[i] = static_cast<uint32_t>(floor(container[i] / h));
    }
  };

  // Calls the neighbor search
  neighborSearch(
    list,
    d_objs_,
    list_size, 
    d_lookup_,
    n_partitions,
    n_particles,
    containerCount,
    h
  );

  // Second verlet pass with new force values
  secondVerletKernel<<<blocks, threads>>>(d_objs_, n_particles);
  hipDeviceSynchronize();
#ifdef __debug
  std::cout << "Second Verlet Pass\n";
#endif
  // If first iteration create managed malloc calls for cpu copy of positions and densities
  if ((*u_positions) == NULL) {
    hipMallocManaged(u_positions, n_particles * sizeof(float) * 3);

    err = hipGetLastError();
    if (err != hipSuccess) {
      std::cerr << "Position Malloc Error: " << hipGetErrorString(err) << '\n';
      exit(EXIT_FAILURE);
    }
  }

  if ((*u_densities) == NULL) {
    hipMallocManaged(u_densities, n_particles * sizeof(float));

    err = hipGetLastError();
    if (err != hipSuccess) {
      std::cerr << "Density Malloc Error: " << hipGetErrorString(err) << '\n';
      exit(EXIT_FAILURE);
    }
  }

  // Creates single contiguous buffer of positions and densities on cpu 
  updateHostBuffer<<<blocks, threads>>>(d_objs_, (*u_positions), (*u_densities), n_particles);

  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "Launch Error: " << hipGetErrorString(err) << '\n';
    exit(EXIT_FAILURE);
  }

  hipDeviceSynchronize();
  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "Sync Error: " << hipGetErrorString(err) << '\n';
    exit(EXIT_FAILURE);
  }
#ifdef __debug
  std::cout << "Filled host buffer\n";
#endif
}
