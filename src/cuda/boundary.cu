#include "hip/hip_runtime.h"
#include "boundary.hpp"
#include "spatial.hpp"

#define tol 1e-4

__host__ static inline bool equal(std::vector<float> new_container, std::vector<float> boundary) {
  for (int i = 0; i < 3; ++i) {
    if (new_container[i] != boundary[i]) return false;
  }
  return true; 
} 

/*
   Updates the bounds given a new container vector. This stalls the program until
   it can accurately update the spatial hashmap for a new size. 
   */
__host__ void updateBounds(Lookup *d_lookup_, particleContainer *d_objs_, std::vector<float> new_container, uint32_t particle_recount, const float h) {
  // refac cuda mem manage now pass cpu resize 
  static std::vector<float> boundary; 
  static uint32_t n_particles  = 0;
  static uint32_t n_partitions = 0;

  bool setLookup = false;

  // Set static size variables

  // Set boundary size
  if (boundary.size() == 0) {
    boundary = std::vector<float>(3, 0);
    boundary = new_container;
  }

  // Set partition count
  if (!n_partitions) {
    // Count the number of partitions ("volume")
    uint32_t partition_counter[3];
    n_partitions = 1;
    for (int i = 0; i < 3; ++i) {
      partition_counter[i] = static_cast<uint32_t>(float(boundary[i] / h));
      n_partitions *= partition_counter[i];
    }

  }

  // Set particle count
  if (!n_particles) {
    n_particles = particle_recount;
  } 

  // Resize particle related structs 
  if (particle_recount != n_particles) {

    // Change lookup table size
    if (findSquare(particle_recount) != findSquare(n_particles))
      setLookup = true;
    
    float minimum = 1.0;
    float maximum = 9.0;  // Hardcoded for now

    // We can pull the position and velocity vectors and then delete everything else and create a new ptr
    float *n_pos, *n_vel;
    
    // Create new memory 
    hipMallocManaged(&n_pos, particle_recount * 3 * sizeof(float));
    hipMallocManaged(&n_vel, particle_recount * 3 * sizeof(float));
    
    // Set copy size
    uint32_t copy = (n_particles < particle_recount) ? n_particles : particle_recount;

    hipMemcpy(n_pos, d_objs_->positions, copy * 3 * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(n_vel, d_objs_->velocities, copy * 3 * sizeof(float), hipMemcpyDeviceToHost);

    delete (d_objs_);

    float *n_prf, *n_visf, *n_mass, *n_dens, *n_pr;

    new (d_objs_) particleContainer();

    // Allocate accumulators
    hipMallocManaged(&n_prf, particle_recount * 3 * sizeof(float));
    hipMallocManaged(&n_visf, particle_recount * 3 * sizeof(float));
    hipMallocManaged(&n_mass, particle_recount * sizeof(float));
    hipMallocManaged(&n_dens, particle_recount * sizeof(float));
    hipMallocManaged(&n_pr, particle_recount * sizeof(float));

    // Add new particles if size is larger
    if (particle_recount > n_particles) {
      d_objs_->addNewParticles(
        n_pos,
        n_vel,
        n_particles, 
        particle_recount,
        minimum,
        maximum
      );
    }

    d_objs_->slowSetAccumulators(
      n_prf,
      n_visf,
      n_mass,
      n_dens,
      n_pr,
      particle_recount
    );
  }

  // Resize container related structs
  if (!equal(new_container, boundary) || setLookup) {
    // Not worrying about it right now
  }
}
/*
   Kernel call to naively, individually rectify potential out of bounds behavior for a particle 
   */
__global__ static void boundaryKernel(const struct Container boundary, particleContainer *d_objs_, uint32_t n_particles, int32_t n_partitions, const float abs_radius) {
uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n_particles) return;
  const float restitution = 0.8;
  
  // Overlap values
  float upperDistance[3], lowerDistance[3];
  
  for (int i = 0; i < 3; ++i) {
    uint32_t co = idx + i * n_particles;

    upperDistance[i] = boundary.upper[i] - d_objs_->positions[co] - abs_radius;
    lowerDistance[i] = d_objs_->positions[co] + abs_radius - boundary.lower[i];

    // Rectify overlap 
    if (upperDistance[i] < abs_radius + tol) {
      // Adjust for upper bounds overlap
      d_objs_->velocities[co] *= -restitution;
      d_objs_->positions[co] -= upperDistance[i];
    } else if (lowerDistance[i] < abs_radius + tol) {
      // Adjust for lower bounds overlap
      d_objs_->velocities[co] *= -restitution; 
      d_objs_->positions[co] += lowerDistance[i];
    }
  }
}

/*
   Host call to handle the call to boundaryKernel 
   Sets the range of acceptable thread idx to act on particles 
   */
__host__ void callToBoundaryConditions(struct Container boundary, particleContainer *d_objs_, uint32_t n_particles, uint32_t n_partitions, const float h) {

  static uint32_t blocks = 0, threads = 0;
  setGridSize(&blocks, &threads, n_particles);

  // Iterate over each container and call kernel for each one 
  boundaryKernel<<<blocks, threads>>>(
    boundary,
    d_objs_,
    n_particles,
    n_partitions,
    h * 0.2   // Absolute radius
  );    // First time d_particles_ is called by device -> Must be migrated to device
  hipDeviceSynchronize();
}
