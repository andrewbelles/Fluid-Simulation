#include "hip/hip_runtime.h"
#include "integrate.hpp"

// Defines 
#define k 3000
#define rho0 1000
#define dt 1e-3       // Change to dynamically shift in value 
#define viscosity 1e-2

// Offset table for kernel 
__constant__ int3 offset_table[27];
static float max_neighbors;

__device__ static inline float magnitude(const float3 a) {
  return sqrtf(a.x * a.x + a.y * a.y + a.z * a.z);
}

/* float3 overloads */

__host__ __device__ static inline float3 operator/(const float3 a, const float val) {
  return make_float3(a.x / val, a.y / val, a.z / val);
}

__host__ __device__ static inline float3 operator*(const float3 a, const float val) {
  return make_float3(a.x * val, a.y * val, a.z * val);
}

__host__ __device__ static inline void operator+=(float3 &a, float3 b) {
  float *vec_a[] = {&a.x, &a.y, &a.z};
  float *vec_b[] = {&b.x, &b.y, &b.z};

  for (int i = 0; i < 3; ++i)
    *vec_a[i] += *vec_b[i];
}

__host__ __device__ static inline float3 operator-(const float3 a, const float3 b) {
  return make_float3(a.x - b.z, a.y - b.z, a.z - b.z);
}

__host__ __device__ static inline void operator-=(float3 &a, float3 b) {
  float *vec_a[] = {&a.x, &a.y, &a.z};
  float *vec_b[] = {&b.x, &b.y, &b.z};

  for (int i = 0; i < 3; ++i)
    *vec_a[i] -= *vec_b[i];
}

/**
 * Cubic Spline smooth field approximating kernel. 
 */
__host__ __device__ float cubicSpline(float distance, float smooth_radius) {
  // Constant values
  const float  q = distance / smooth_radius;
  const float  a3 = 1.0 / (M_PI * smooth_radius * smooth_radius * smooth_radius);
  float value = a3;

  // Calcuate value of kernel over the smoothing radius
  if (q >= 0 && q < 1) {
    value *= (1.0 - (1.5 * q * q) + 0.75 * q * q * q);
  } else if (q >= 1 && q < 2) {
    value *= (0.25 * (2.0 - q) * (2.0 - q) * (2.0 - q)); 
  // Outside influence
  } else if (q >= 2) {
    value = 0;
  }
  return value;
}

/*
   The gradient of the Cubic Spline kernel 
   */
__host__ __device__ float gradCubicSpline(float distance, float smooth_radius) {
  // Constant values
  const float  q = distance / smooth_radius;
  const float  a3 = 1.0 / (M_PI * smooth_radius * smooth_radius * smooth_radius);
  float value = a3;

  // Calculate the gradient of the kernel over the smoothing radius
  if (q >= 0 && q < 1) {
    value *= (-3.0 * q + 2.25 * q * q);
  } else if (q >= 1 && q < 2) {
    value *= (-0.75 * (2.0 - q) * (2.0 - q));
  // Outside influence
  } else if (q >= 2) {
    value = 0;
  }
  return value;
}

/*
   The laplacian of the Cubic Spline kernel 
   */
__host__ __device__ float laplacianCubicSpline(float distance, float smooth_radius) {
  const float  q = distance / smooth_radius;
  const float  a3 = 1.0 / (M_PI * smooth_radius * smooth_radius * smooth_radius);
  float value = a3;

  // Calculate the laplacian of the kernel over the smoothing radius
  if (q >= 0 && q < 1) {
    value *= (-3.0 + 4.5 * q);
  } else if (q >= 1 && q < 2) {
    value *= (1.5 * (2.0 - q));
  // Outside incluence
  } else if (q >= 2) {
    value = 0;
  }
  return value;
}

/**
 * Create the offset table and copy symbol to GPU memory 
 */ 
__host__ void initOffsetTable() {
  int3 host_offset[27];

  int idx = 0;
  // Iterate over 3x3x3 grid 
  for (int dz = -1; dz <= 1; ++dz) {
    for (int dy = -1; dy <= 1; ++dy) {
      for (int dx = -1; dx <= 1; ++dx) {
        // printf("%d %d %d\n", dz, dy, dx);
        host_offset[idx++] = make_int3(dx, dy, dz);  
      }
    }
  }

  // Copy the host table to the global constant offset table
  hipMemcpyToSymbol(HIP_SYMBOL(offset_table), host_offset, sizeof(host_offset));
}

__device__ static inline bool inBounds(const int3 a, const uint32_t bounds[3]) {
  if (a.x >= bounds[0] || a.x < 0 || a.y >= bounds[1] || a.y < 0 || a.z >= bounds[2] || a.z < 0)
    return false;
  else
    return true;
}

/**
 * Operator for int3 adding
 */
__device__ static inline int3 operator+(int3 a, int3 b) {
  return make_int3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__host__ neighborList *initNeighborList(uint32_t *list_size, uint32_t n_particles) {
  max_neighbors = (n_particles * 0.5);
  (*list_size) = static_cast<uint32_t>(n_particles * (max_neighbors - 1));

  neighborList *list = nullptr;

  hipMallocManaged(&list->neighbors, (*list_size) * sizeof(int));
  hipMallocManaged(&list->offsets, (n_particles + 1) * sizeof(int));
  hipMallocManaged(&list->counts, n_particles * sizeof(int));

  hipMallocManaged(&list, sizeof(neighborList));

  return list;
}

/*
 * Iterate over 3x3x3 centered by idx and determine number of neighbors for particle
 */
__global__ static void countNeighbors(
  neighborList *list,
  particleContainer *d_objs_,
  uint32_t list_size,
  Lookup *d_lookup_,
  uint32_t n_partitions,
  uint32_t n_particles,
  uint32_t containerCount[3],
  float h
) {

  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n_particles) return; 
  
  uint32_t pid = d_lookup_->table_[idx].idx;
  uint32_t rel = 0, hash = 0, start = 0, end = 0;
  int3 cell_coord, relative_coord;

  float3 relative_pos, displace;
  float3 local_pos = make_float3(
    d_objs_->positions[pid],
    d_objs_->positions[pid + n_particles],
    d_objs_->positions[pid + 2 * n_particles]
  );

  float distance = 0.0;

  // Find cell coordinate from pid position
  cell_coord = positionToCellCoord(local_pos, h);

  for (int i = 0; i < 27; ++i) {
    relative_coord = cell_coord + offset_table[i];

    // Check if in bounds 
    if (!inBounds(relative_coord, containerCount)) continue;

    // Hash relative coordinate and get start and end values 
    hash  = hashPosition(relative_coord, n_partitions);
    start = d_lookup_->start_cell[hash];
    end   = d_lookup_->end_cell[hash];

    // Empty bucket check
    if (start == UINT32_MAX || end == UINT32_MAX) continue;

    for (uint32_t j = start; j < end; ++j) {

      rel = d_lookup_->table_[j].idx;

      relative_pos = make_float3(
        d_objs_->positions[rel],
        d_objs_->positions[rel + n_particles],
        d_objs_->positions[rel + 2 * n_particles]
      );

      // Find distance 
      displace = local_pos - relative_pos;
      distance= magnitude(displace);

      if (distance > 2.0 * h) continue;

      list->counts[idx]++;
    }
  }
}

__global__ static void findNeighbors(
  neighborList *list,
  particleContainer *d_objs_,
  int status,
  uint32_t list_size,
  Lookup *d_lookup_,
  uint32_t n_partitions,
  uint32_t n_particles,
  uint32_t containerCount[3],
  float h
) {

  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n_particles) return; 
  
  uint32_t pid = d_lookup_->table_[idx].idx;
  uint32_t count = 0, rel = 0, hash = 0, start = 0, end = 0;
  int3 cell_coord, relative_coord;

  float3 relative_pos, displace;
  float3 local_pos = make_float3(
    d_objs_->positions[pid],
    d_objs_->positions[pid + n_particles],
    d_objs_->positions[pid + 2 * n_particles]
  );

  float distance = 0.0;

  // Find cell coordinate from pid position
  cell_coord = positionToCellCoord(local_pos, h);

  for (int i = 0; i < 27; ++i) {
    relative_coord = cell_coord + offset_table[i];

    // Check if in bounds 
    if (!inBounds(relative_coord, containerCount)) continue;

    // Hash relative coordinate and get start and end values 
    hash  = hashPosition(relative_coord, n_partitions);
    start = d_lookup_->start_cell[hash];
    end   = d_lookup_->end_cell[hash];

    // Empty bucket check
    if (start == UINT32_MAX || end == UINT32_MAX) continue;

    for (uint32_t j = start; j < end; ++j) {

      rel = d_lookup_->table_[j].idx;

      relative_pos = make_float3(
        d_objs_->positions[rel],
        d_objs_->positions[rel + n_particles],
        d_objs_->positions[rel + 2 * n_particles]
      );

      // Find distance 
      displace = local_pos - relative_pos;
      distance= magnitude(displace);

      if (distance > 2.0 * h) continue;
      
      int ptr = list->offsets[idx] + count++;
      if (ptr >= list_size) {
        atomicExch(&status, 1);
        return;
      }

      list->neighbors[ptr] = rel; 
    }
  }
}

/**
 * Find the density, pressure, and system forces from the built neighbor list
 */
__global__ static void computeDensities(
  neighborList *list,
  particleContainer *d_objs_,
  uint32_t n_particles,
  float h
) { 
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n_particles) return; 

  // In this function idx == pid therefore idxth value of neighbors 

  float3 local_pos = make_float3(
    d_objs_->positions[idx],
    d_objs_->positions[idx + n_particles],
    d_objs_->positions[idx + 2 * n_particles]
  );

  for (int i = list->offsets[idx]; i < list->offsets[idx + 1]; ++i) {

    uint32_t rel = list->neighbors[i];

    float3 relative_pos = make_float3(
      d_objs_->positions[rel],
      d_objs_->positions[rel + n_particles],
      d_objs_->positions[rel + 2 * n_particles] 
    );

    // Find distance
    float3 displace = local_pos - relative_pos;
    float distance  = magnitude(displace);

    // Find density sum
    d_objs_->densities[idx] += d_objs_->masses[rel] * cubicSpline(distance, h);
  }
  d_objs_->pressures[idx] = k * (d_objs_->densities[idx] - rho0);
}

__global__ static void computeForces(
  neighborList *list,
  particleContainer *d_objs_,
  uint32_t n_particles,
  const float h
) {
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n_particles) return; 

  // Set local values
  float3 pressure_force  = make_float3(0.0, 0.0, 0.0);
  float3 viscosity_force = make_float3(0.0, 0.0, 0.0);

  float3 local_pos = make_float3(
    d_objs_->positions[idx],
    d_objs_->positions[idx + n_particles],
    d_objs_->positions[idx + 2 * n_particles]
  );
 
  float3 local_vel = make_float3(
    d_objs_->velocities[idx],
    d_objs_->velocities[idx + n_particles],
    d_objs_->velocities[idx + 2 * n_particles]
  );


  // Loop over neighbor indexes
  for (uint32_t i = list->offsets[idx]; i < list->offsets[idx + 1]; ++i) {
    // Set the relative particle id
    uint32_t rel = list->neighbors[i];
    
    float3 relative_pos = make_float3(
      d_objs_->positions[rel],
      d_objs_->positions[rel + n_particles],
      d_objs_->positions[rel + 2 * n_particles] 
    );

    // Find distance
    float3 displace = local_pos - relative_pos;
    float distance  = magnitude(displace);

    float3 relative_vel = make_float3( 
      d_objs_->velocities[rel],
      d_objs_->velocities[rel + n_particles],
      d_objs_->velocities[rel + 2 * n_particles]
    );

    // Unit vector of direction calculatio 
    float3 direction = displace / (distance + tol);

    // Calculate the intermediate values for the pressure force;
    float pressure_value  = d_objs_->pressures[idx] / (d_objs_->densities[idx] * d_objs_->densities[idx]);
    pressure_value       += d_objs_->pressures[rel] / (d_objs_->densities[rel] * d_objs_->densities[rel]); 
    float common_term    = d_objs_->masses[rel] * pressure_value * gradCubicSpline(distance, h);

    pressure_force  -= (direction * common_term);  
    viscosity_force += (((local_vel - relative_vel) * d_objs_->masses[rel] / d_objs_->densities[rel]) * laplacianCubicSpline(distance, h));
  }

  // Set forces 
  d_objs_->pressure_forces[idx] = pressure_force.x;
  d_objs_->pressure_forces[idx + n_particles] = pressure_force.y;
  d_objs_->pressure_forces[idx + 2 * n_particles] = pressure_force.z;
 
  d_objs_->viscosity_forces[idx] = viscosity_force.x;
  d_objs_->viscosity_forces[idx + n_particles] = viscosity_force.y;
  d_objs_->viscosity_forces[idx + 2 * n_particles] = viscosity_force.z;
}

__global__ static void resetCounts(int *counts, uint32_t n_particles) {
  for (int i = 0; i < n_particles; i++) {
    counts[i] = 0;
  }
}

/**
 * Host function to call the search kernel to find each particles forces relative to itself
 */
__host__ void neighborSearch(
  neighborList *list,
  particleContainer *d_objs_,
  Lookup *d_lookup_,
  uint32_t n_partitions, 
  uint32_t n_particles,
  uint32_t containerCount[3],
  uint32_t *list_size,
  float h
) {
  static uint32_t blocks = 0, threads = 0;
  setGridSize(&blocks, &threads, n_particles);
  int status = 0;
  hipError_t err;

  // Perform computation until neighbor list is valid sized 
  do {
    countNeighbors<<<blocks, threads>>>(
      list,
      d_objs_,
      (*list_size), 
      d_lookup_,
      n_partitions,
      n_particles,
      containerCount,
      h
    );

    // Find prefix sum of list stored at offsets 
    thrust::exclusive_scan(thrust::device, list->counts, list->counts + n_particles, list->offsets);

    findNeighbors<<<blocks, threads>>>(
      list,
      d_objs_,
      status,
      (*list_size), 
      d_lookup_,
      n_partitions,
      n_particles,
      containerCount,
      h
    );
    
    err = hipGetLastError();
    if (err != hipSuccess) {
      std::cerr << "Neighbor List: " << hipGetErrorString(err) << '\n';
      exit(EXIT_FAILURE);
    }

    // Wait for all threads to complete before restarting if size error
    hipDeviceSynchronize();
   
    err = hipGetLastError();
    if (err != hipSuccess) {
      std::cerr << "List sync: " << hipGetErrorString(err) << '\n';
      exit(EXIT_FAILURE);
    }
    
    // Neighbors list size error
    if (status == 1) {
      // Take truncated value of 3/2 k and resize neighbors list
      max_neighbors = max_neighbors / n_particles + max_neighbors * 0.5;
      (*list_size) = static_cast<uint32_t>(n_particles * (max_neighbors - 1));
      hipFree(list->neighbors);
      hipMallocManaged(&list->neighbors, (*list_size) * n_particles * sizeof(uint32_t));
    }

  } while (status != 0); 

  computeDensities<<<blocks, threads>>>(
    list,
    d_objs_,
    n_particles,
    h
  );

  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "Force Compute: " << hipGetErrorString(err) << '\n';
    exit(EXIT_FAILURE);
  }
  
  hipDeviceSynchronize();

  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "Force Sync: " << hipGetErrorString(err) << '\n';
    exit(EXIT_FAILURE);
  }

  // Expected success 
  computeForces<<<blocks, threads>>>(
    list,
    d_objs_,
    n_particles,
    h
  );
  
  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "Force Compute: " << hipGetErrorString(err) << '\n';
    exit(EXIT_FAILURE);
  }
  
  hipDeviceSynchronize();

  err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "Force Sync: " << hipGetErrorString(err) << '\n';
    exit(EXIT_FAILURE);
  }

  resetCounts<<<1, 1>>>(list->counts, n_particles);
}

/*
   Completes the first section of verlet integration
   */
__global__ void firstVerletKernel(particleContainer *d_objs_, uint32_t n_particles) {
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n_particles) return;

  float forceSum[3] = {0.0, -9.81, 0.0};

  // Position and velocity update loop
  for (int i = 0; i < 3; ++i) {
    uint32_t co = idx + i * n_particles;
   
    // Sums the pressure and viscosity forces for each axis
    forceSum[i] += (d_objs_->pressure_forces[co] + d_objs_->viscosity_forces[co]);
    
    // Integrates the velocity and position
    d_objs_->velocities[co] += (forceSum[i] * static_cast<float>(0.5 * dt));
    d_objs_->positions[co] += (d_objs_->velocities[co] * static_cast<float>(dt));
  }
}

/*
   Second pass of verlet integration
   */
__global__ void secondVerletKernel(particleContainer *d_objs_, uint32_t n_particles) {
  uint32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= n_particles) return;

  float forceSum[3] = {0.0, -9.81, 0.0};

  // iterate over axis
  for (int i = 0; i < 3; ++i) {
    uint32_t co = idx + i * n_particles;
    // Sum forces from previous iteration 
    forceSum[i] += ((d_objs_->pressure_forces[co] + d_objs_->viscosity_forces[co]) / d_objs_->masses[idx]);
    
    // Second half step to fully velocity
    d_objs_->velocities[co] += (forceSum[i] * static_cast<float>(0.5 * dt));
  }
}
